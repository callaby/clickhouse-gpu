#include "hip/hip_runtime.h"
// Copyright 2016-2020 NVIDIA
//    Licensed under the Apache License, Version 2.0 (the "License");
//    you may not use this file except in compliance with the License.
//    You may obtain a copy of the License at
//        http://www.apache.org/licenses/LICENSE-2.0
//    Unless required by applicable law or agreed to in writing, software
//    distributed under the License is distributed on an "AS IS" BASIS,
//    WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//    See the License for the specific language governing permissions and
//    limitations under the License.

#include <Common/Cuda/CudaAtomics.cuh>
#include <Common/Cuda/CudaSafeCall.h>

#include <AggregateFunctions/Cuda/CudaAggregateFunctionCount.h>

namespace DB
{

__global__ void  kerCudaAddBulkCount(CudaAggregateFunctionCountData *places, 
    ICudaAggregateFunction::CudaSizeType elements_num, 
    ICudaAggregateFunction::CudaSizeType *res_buckets)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < elements_num)) return;
    ICudaAggregateFunction::CudaSizeType    res_bucket = res_buckets[i];
    cuda_details::atomicAdd(&(places[res_bucket].count), (UInt64)1);
}


__global__ void  kerCudaMergeBulkCount(CudaAggregateFunctionCountData *places, ICudaAggregateFunction::CudaSizeType elements_num,
    CudaAggregateFunctionCountData *places_from, ICudaAggregateFunction::CudaSizeType *res_buckets)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < elements_num)) return;
    ICudaAggregateFunction::CudaSizeType    res_bucket = res_buckets[i];
    if (res_bucket == ~((ICudaAggregateFunction::CudaSizeType)0)) return;
    cuda_details::atomicAdd(&(places[res_bucket].count), places_from[i].count);
}


void CudaAggregateFunctionCount::cudaAddBulk(CudaAggregateDataPtr places, CudaColumnStringPtr str_column,
    CudaSizeType elements_num, CudaSizeType *res_buckets, 
    char *tmp_buf, hipStream_t stream) const
{
    /// TODO Strange unused parameter error
    tmp_buf = tmp_buf; str_column = str_column;
    kerCudaAddBulkCount<<<(elements_num/256)+1,256,0,stream>>>(
        (CudaAggregateFunctionCountData*)places, elements_num, res_buckets);
}


void CudaAggregateFunctionCount::cudaMergeBulk(CudaAggregateDataPtr places, CudaSizeType elements_num,
    CudaAggregateDataPtr places_from, CudaSizeType *res_buckets, 
    hipStream_t stream) const
{
    kerCudaMergeBulkCount<<<(elements_num/256)+1,256,0,stream>>>(
        (CudaAggregateFunctionCountData*)places, elements_num, 
        (CudaAggregateFunctionCountData*)places_from, res_buckets);
}

/*class CudaAggregateFunctionCount final : public ICudaAggregateFunction
{
public:

    ResultType  getResult(AggregateDataPtr place) const override
    {
        return ((CudaAggregateFunctionCountData*)place)->count;
    }
};*/


}
