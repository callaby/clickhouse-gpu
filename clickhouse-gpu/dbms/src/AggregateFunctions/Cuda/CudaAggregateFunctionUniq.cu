#include "hip/hip_runtime.h"
// Copyright 2016-2020 NVIDIA
//    Licensed under the Apache License, Version 2.0 (the "License");
//    you may not use this file except in compliance with the License.
//    You may obtain a copy of the License at
//        http://www.apache.org/licenses/LICENSE-2.0
//    Unless required by applicable law or agreed to in writing, software
//    distributed under the License is distributed on an "AS IS" BASIS,
//    WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//    See the License for the specific language governing permissions and
//    limitations under the License.

#include <Common/Cuda/CudaAtomics.cuh>
//#include <Common/Cuda/City_Hash/city.h>
#include <Common/Cuda/cudaCalcMurmurHash64.h>
#include <Common/Cuda/cudaCalcCityHash64.h>

#include <AggregateFunctions/Cuda/CudaAggregateFunctionUniq.h>


namespace DB
{

/// the only supported 'type'(T) is String

__global__ void  kerCudaInitAggregateData(ICudaAggregateFunction::CudaSizeType places_num,
    CudaAggregateFunctionUniqHLL12Data *places)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < places_num)) return;
    (places + i)->initNonzeroData();
    //new (places + i) CudaAggregateFunctionUniqHLL12Data();
}

__global__ void  kerCudaAddBulk(CudaAggregateFunctionUniqHLL12Data *places, 
        ICudaAggregateFunction::CudaSizeType elements_num, const UInt64 *hashes, 
        ICudaAggregateFunction::CudaSizeType *res_buckets)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < elements_num)) return;
    UInt64                              hash = hashes[i];
    ICudaAggregateFunction::CudaSizeType    res_bucket = res_buckets[i];
    places[res_bucket].set.insert(hash);
}

__global__ void  kerCudaMergeBulk(CudaAggregateFunctionUniqHLL12Data *places, ICudaAggregateFunction::CudaSizeType elements_num,
        CudaAggregateFunctionUniqHLL12Data *places_from, ICudaAggregateFunction::CudaSizeType *res_buckets)
{
    ICudaAggregateFunction::CudaSizeType i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < elements_num)) return;
    ICudaAggregateFunction::CudaSizeType    res_bucket = res_buckets[i];
    if (res_bucket == ~((ICudaAggregateFunction::CudaSizeType)0)) return;
    places[res_bucket].set.merge(places_from[i].set);
}

/// The only supported Data here is CudaAggregateFunctionUniqHLL12Data

void CudaAggregateFunctionUniq<String, CudaAggregateFunctionUniqHLL12Data>::cudaInitAggregateData(
    CudaSizeType places_num, CudaAggregateDataPtr places, hipStream_t stream) const
{
    CUDA_SAFE_CALL( hipMemset( places, 0, sizeof(CudaAggregateFunctionUniqHLL12Data)*places_num ) );
    kerCudaInitAggregateData<<<(places_num/256)+1,256,0,stream>>>(places_num, (CudaAggregateFunctionUniqHLL12Data*)places);
}

void CudaAggregateFunctionUniq<String, CudaAggregateFunctionUniqHLL12Data>::cudaAddBulk(
    CudaAggregateDataPtr places, CudaColumnStringPtr str_column,
    CudaSizeType elements_num, CudaSizeType *res_buckets, 
    char *tmp_buf, hipStream_t stream) const
{
    cudaCalcCityHash64(elements_num, str_column->getBuf(), false, str_column->getLens(), 
        str_column->getOffsets(), (UInt64*)tmp_buf, stream);

    kerCudaAddBulk<<<(elements_num/256)+1,256,0,stream>>>(
        (CudaAggregateFunctionUniqHLL12Data*)places, elements_num, (UInt64*)tmp_buf, res_buckets);
}

void CudaAggregateFunctionUniq<String, CudaAggregateFunctionUniqHLL12Data>::cudaMergeBulk(
    CudaAggregateDataPtr places, CudaSizeType elements_num,
    CudaAggregateDataPtr places_from, CudaSizeType *res_buckets, 
    hipStream_t stream) const
{
    kerCudaMergeBulk<<<(elements_num/256)+1,256,0,stream>>>(
        (CudaAggregateFunctionUniqHLL12Data*)places, elements_num, 
        (CudaAggregateFunctionUniqHLL12Data*)places_from, res_buckets);
}

}
