#include "hip/hip_runtime.h"
#include <stdexcept>
#include <cassert>
//#include <hipcub/hipcub.hpp>

#include <Common/Cuda/CudaSafeCall.h>

#include <Columns/Cuda/CudaColumnString.h>


namespace DB
{

CudaColumnString::CudaColumnString(size_t max_str_num_,size_t max_sz_) : max_str_num(max_str_num_), max_sz(max_sz_)
{
    if ((max_str_num_ == 0)||(max_sz_ == 0)) throw std::logic_error("CudaColumnString: try to create zero size buffer");
    hipError_t err;
    err = hipMalloc( (void**)&buf, max_sz*sizeof(char) );
    if (err != hipSuccess) throw std::runtime_error("CudaColumnString: failed to alloc cuda memory for strings");
    err = hipMalloc( (void**)&lens, max_str_num*sizeof(UInt32) );
    if (err != hipSuccess) {
        CUDA_SAFE_CALL_NOTHROW( hipFree(buf) );
        throw std::runtime_error("CudaColumnString: failed to alloc cuda memory for lengths buffer");
    }
    err = hipMalloc( (void**)&offsets, max_str_num*sizeof(UInt32) );
    if (err != hipSuccess) {
        CUDA_SAFE_CALL_NOTHROW( hipFree(buf) ); 
        CUDA_SAFE_CALL_NOTHROW( hipFree(lens) );
        throw std::runtime_error("CudaColumnString: failed to alloc cuda memory for offsets buffer");
    }
    /*buf4_sz = 0;
    hipcub::DeviceScan::ExclusiveSum(nullptr, buf4_sz, lens, offsets, max_str_num);
    err = hipMalloc( (void**)&tmp_buf4, buf4_sz );*/
    err = hipMalloc( (void**)&offsets64, max_str_num*sizeof(UInt64) );
    if (err != hipSuccess) {
        CUDA_SAFE_CALL_NOTHROW( hipFree(buf) ); 
        CUDA_SAFE_CALL_NOTHROW( hipFree(lens) ); 
        CUDA_SAFE_CALL_NOTHROW( hipFree(offsets) ); 
        //throw std::runtime_error("CudaColumnString: failed to alloc cuda memory for additional buffer");
        throw std::runtime_error("CudaColumnString: failed to alloc cuda memory for offsets64 buffer");
    }
}


void CudaColumnString::setSize(size_t str_num_, size_t sz_)
{
    str_num = str_num_; sz = sz_;
}

__global__ void kerCalcLengths(UInt32 block_begin, UInt32 block_size, 
    UInt32 block_offset, UInt32 block_buf_size, UInt64 *offsets64, 
    UInt32 *lens, UInt32 *offsets)
{
    UInt32  i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!(i < block_size)) return;   
    UInt32  local_offset,
            local_offset_next = offsets64[block_begin + i];
    if (i > 0) 
        local_offset = offsets64[block_begin + i-1];
    else
        local_offset = 0;

    offsets[block_begin + i] = local_offset + block_offset;
    lens[block_begin + i] = local_offset_next - local_offset;
    //offsets[block_begin + i] = block_begin + i;
    //lens[block_begin + i] = 1;
}

void CudaColumnString::calcLengths(const std::vector<UInt32> &blocks_sizes, 
    const std::vector<UInt32> &blocks_buf_sizes, hipStream_t stream)
{
    assert(blocks_sizes.size() == blocks_buf_sizes.size());
    UInt32  block_begin = 0, block_offset = 0;
    for (size_t i = 0;i < blocks_sizes.size();++i) 
    {
        kerCalcLengths<<<(blocks_sizes[i]/256)+1,256,0,stream>>>(
            block_begin, blocks_sizes[i], block_offset, blocks_buf_sizes[i], 
            offsets64, lens, offsets);
        block_begin += blocks_sizes[i]; 
        block_offset += blocks_buf_sizes[i];
    }
}


CudaColumnString::~CudaColumnString()
{
    CUDA_SAFE_CALL_NOTHROW( hipFree(buf) );
    CUDA_SAFE_CALL_NOTHROW( hipFree(lens) );
    CUDA_SAFE_CALL_NOTHROW( hipFree(offsets) );
    CUDA_SAFE_CALL_NOTHROW( hipFree(offsets64) );
    //CUDA_SAFE_CALL_NOTHROW( hipFree(tmp_buf4) );
}

}
